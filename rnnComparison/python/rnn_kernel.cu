#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <vector>


#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


std::vector<torch::Tensor> rnn_cuda_forward(
    torch::Tensor input,
    torch::Tensor weights,
    torch::Tensor bias,
    torch::Tensor init_h) {

  hipStream_t inputStream;
  hipStreamCreate(&inputStream);
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetStream(handle, inputStream);
  int batch = input.sizes()[0];
  int sequence_length = input.sizes()[1];
  int input_size = input.sizes()[2];
  int hidden_size = weights.sizes()[1];
  float alpha = 1.f;
  float beta = 0.f;
  const auto options = input.options();
  torch::Tensor output = torch::empty({batch, sequence_length, hidden_size}, options);

  
  // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
  //     sequence_length, hidden_size, input_size, 
  //     &alpha, (float *)input.data_ptr(), sequence_length,
  //     (float *)weights.data_ptr(), input_size, &beta, output.data_ptr<float>(), sequence_length);
  hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
      sequence_length, hidden_size, input_size, 
      &alpha, input.data_ptr<float>(), sequence_length, sequence_length*input_size,
      weights.data_ptr<float>(), input_size, 0, &beta, output.data_ptr<float>(), sequence_length, sequence_length*hidden_size, 1);
  

  return {output};
}

